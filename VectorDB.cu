#include "hip/hip_runtime.h"
#include <cstdio>

#include "rocksdb/db.h"
#include "rocksdb/options.h"
#include "rocksdb/slice.h"

#include <thrust/sequence.h>

#include "kNearestNeighbors.h"

using ROCKSDB_NAMESPACE::DB;
using ROCKSDB_NAMESPACE::Iterator;
using ROCKSDB_NAMESPACE::Options;
using ROCKSDB_NAMESPACE::PinnableSlice;
using ROCKSDB_NAMESPACE::ReadOptions;
using ROCKSDB_NAMESPACE::Status;
using ROCKSDB_NAMESPACE::WriteBatch;
using ROCKSDB_NAMESPACE::WriteOptions;

// Design decision: Seperate deviceKeys and vectorKeys.
//
// vectorKeys can be very long (e.g., a whole UUID) but deviceKeys must
// be a single unsigned integer (e.g., 32 bits). This imposes a limitation
// that at most 2^32 billion vectors can be searched on a GPU at a time, no
// matter the memory constraints of the GPU. This can be changed in the future.
//
// a separate collection of deviceKey-to-vectorkey key-value pairs will be kept
// in either CPU memory or the persistent key value store (e.g., RocksDB).
//
// in addition, deviceKeys will be sequential in order to enable quick vector
// retrieval by interpreting them as indexes in the on-device vector array

// Requires keys to be sequential, representing array indexes
__global__ void retrieveVectorsFromKeys(uint64_cu *vectors, unsigned *keys,
                                        int numKeys, uint64_cu *retrieved) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;

  for (int i = idx; i < numKeys; i += stride)
    retrieved[i] = vectors[keys[i]];
}

void printBits(uint64_cu &x) {
  std::bitset<sizeof(uint64_cu) * CHAR_BIT> b(x);
  std::cout << b << std::endl;
}

class VectorDB {
private:
  // Pointers to device memory
  unsigned *workingMem1;
  unsigned *workingMem2;
  unsigned *workingMem3;
  uint64_cu *vectors;
  uint64_cu *deviceQueryVector;
  unsigned *deviceKeys; // sequential keys

  // Use RocksDB as persistent key-value store
  rocksdb::DB *db;

public:
  int numVectors;

  VectorDB(const std::string &name, int capacity) {
    // ROCKSDB INITIALIZATION
    // Open key value store or create it if it doesn't exist
    Options options;
    options.create_if_missing = true;
    Status s = DB::Open(options, name, &db);
    assert(s.ok());

    // Retrieve numVectors variable if it exists or initialize it doesn't
    std::string value;
    s = db->Get(ReadOptions(), "numVectors", &value);
    if (s.IsNotFound()) {
      s = db->Put(WriteOptions(), "numVectors", std::to_string(0));
      assert(s.ok());
      numVectors = 0;
    } else {
      assert(s.ok());
      numVectors = std::stoi(value);
    }

    // CUDA INITIALIZATION
    // Allocate all on-device memory
    hipMalloc(&workingMem1, capacity * sizeof(unsigned));
    hipMalloc(&workingMem2, capacity * sizeof(unsigned));
    hipMalloc(&workingMem3, capacity * sizeof(unsigned));
    hipMallocManaged(&vectors, capacity * sizeof(uint64_cu));
    hipMallocManaged(&deviceQueryVector, sizeof(uint64_cu));
    hipMalloc(&deviceKeys, capacity * sizeof(unsigned));

    // Initialize device keys
    thrust::sequence(thrust::device, deviceKeys, deviceKeys + capacity);

    // Load vectors from db to device
    int iterCount = 0;
    Iterator *iter = db->NewIterator(ReadOptions());
    iter->SeekToFirst();

    uint64_cu *hostVectors =
        (uint64_cu *)malloc(numVectors * sizeof(uint64_cu));
    for (; iter->Valid(); iter->Next(), ++iterCount) {
      assert(iterCount <= numVectors);

      // TODO: use column families so we don't have to do this kind of checking
      if (iter->key().ToString() != "numVectors") {
        hostVectors[iterCount] = std::stoull(iter->value().ToString());
      }
    }
    delete iter;
    hipMemcpy(vectors, hostVectors, numVectors * sizeof(uint64_cu),
               hipMemcpyHostToDevice);
    free(hostVectors);
  }

  ~VectorDB() {
    delete db;

    hipFree(workingMem1);
    hipFree(workingMem2);
    hipFree(workingMem3);
    hipFree(vectors);
    hipFree(deviceQueryVector);
    hipFree(deviceKeys);
  }

  // void loadDevice() {

  // }

  /*
    Inserts new key. Panics if key already exists
  */
  void insert(const std::string &vectorKey, uint64_cu &vector) {
    // NOTE: These two should eventually be made into a transaction
    // Check if the vectorKey exists
    std::string value;
    Status getStatus = db->Get(ReadOptions(), vectorKey, &value);
    assert(getStatus.IsNotFound());

    // Write to db and device
    Status putStatus =
        db->Put(WriteOptions(), vectorKey, std::to_string(vector));
    assert(putStatus.ok());
    hipMemcpy(vectors + numVectors, &vector, sizeof(uint64_cu),
               hipMemcpyHostToDevice);

    // Update numVectors
    Status s =
        db->Put(WriteOptions(), "numVectors", std::to_string(numVectors + 1));
    assert(s.ok());
    numVectors++;
  }

  void query(uint64_cu *queryVector, int k, float *kNearestDistances,
             uint64_cu *kNearestVectors) {
    float *deviceKNearestDistances;
    unsigned *deviceKNearestKeys;
    uint64_cu *deviceKNearestVectors;
    hipMallocManaged(&deviceKNearestDistances, k * sizeof(float));
    hipMallocManaged(&deviceKNearestKeys, k * sizeof(unsigned));
    hipMalloc(&deviceKNearestVectors, k * sizeof(uint64_cu));

    hipMemcpy(deviceQueryVector, queryVector, sizeof(uint64_cu),
               hipMemcpyHostToDevice);

    kNearestNeighbors(vectors, deviceKeys, deviceQueryVector, numVectors, k,
                      deviceKNearestDistances, deviceKNearestKeys, workingMem1,
                      workingMem2, workingMem3);

    for (int i = 0; i < k; ++i) {
      printf("deviceKNearestKeys: %d: %u\n", i, deviceKNearestKeys[i]);
    }
    // retrieve vectors from relevant keys
    retrieveVectorsFromKeys<<<1, 1024>>>(vectors, deviceKNearestKeys, k,
                                         deviceKNearestVectors);
    hipDeviceSynchronize();

    // copy solution from device to host specified by caller
    hipMemcpy(kNearestDistances, deviceKNearestDistances, k * sizeof(float),
               hipMemcpyDeviceToHost);
    hipMemcpy(kNearestVectors, deviceKNearestVectors, k * sizeof(uint64_cu),
               hipMemcpyDeviceToHost);

    hipFree(deviceKNearestDistances);
    hipFree(deviceKNearestKeys);
    hipFree(deviceKNearestVectors);
  }
};
