#include "hip/hip_runtime.h"
#include <assert.h>
#include <stdio.h>
#include <stdlib.h>

#include "radix_select.h"

__device__ unsigned hash(unsigned a) {
  a = (a ^ 61) ^ (a >> 16);
  a = a + (a << 3);
  a = a ^ (a >> 4);
  a = a * 0x27d4eb2d;
  a = a ^ (a >> 15);
  return a;
}

__global__ void rand(int n, unsigned *xs) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;

  for (int i = idx; i < n; i += stride)
    xs[idx] = hash(~idx);
}

int main() {
  int n = 1 << 30;
  int k = 10;

  int blockSize = 512;
  int numBlocks = (n + blockSize - 1) / blockSize;

  // generate random numbers
  unsigned *xs;
  hipMalloc(&xs, n * sizeof(unsigned));
  rand<<<numBlocks, blockSize>>>(n, xs);
  hipDeviceSynchronize();

  // allocate and initalize keys on device
  unsigned *keys;
  hipMalloc(&keys, n * sizeof(unsigned));
  thrust::sequence(thrust::device, keys, keys + n);

  // allocate kSmallestKeys and kSmallestValues on host
  unsigned *kSmallestKeys = (unsigned *)malloc(k * sizeof(unsigned));
  unsigned *kSmallestValues = (unsigned *)malloc(k * sizeof(unsigned));

  unsigned *tempValues1, *tempValues2;
  hipMalloc(&tempValues1, n * sizeof(unsigned));
  hipMalloc(&tempValues2, n * sizeof(unsigned));


  // run radix select
  float time;
  hipEvent_t start, stop;

  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  radix_select(xs, keys, n, k, kSmallestValues, kSmallestKeys, tempValues1, tempValues2);

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&time, start, stop);

  printf("Execution time:  %.3f ms \n", time);

  for (int i = 0; i < k; ++i) {
    printf("kSmallestKeys: %d: %u\n", i, kSmallestKeys[i]);
  }
  for (int i = 0; i < k; ++i) {
    printf("kSmallestValues: %d: %u\n", i, kSmallestValues[i]);
  }

  // // run thrust sort
  // hipEventCreate(&start);
  // hipEventCreate(&stop);
  // hipEventRecord(start, 0);

  // thrust::sort(thrust::device, xs, xs + n);

  // hipEventRecord(stop, 0);
  // hipEventSynchronize(stop);
  // hipEventElapsedTime(&time, start, stop);

  // printf("Execution time:  %.3f ms \n", time);

  hipFree(xs);
  hipFree(keys);
  hipFree(tempValues1);
  hipFree(tempValues2);

  free(kSmallestKeys);

  return 0;
}