#include "hip/hip_runtime.h"
#include <bitset>
#include <iostream>
#include <math.h>
#include <sys/time.h>

#include <thrust/device_vector.h>
#include <thrust/functional.h>
#include <thrust/generate.h>
#include <thrust/host_vector.h>
#include <thrust/random.h>
#include <thrust/reduce.h>

#include "knn.inl"

// murmur64 hash function
__device__ uint64_cu hash(uint64_cu h) {
  h ^= h >> 33;
  h *= 0xff51afd7ed558ccdL;
  h ^= h >> 33;
  h *= 0xc4ceb9fe1a85ec53L;
  h ^= h >> 33;
  return h;
}

__global__ void randf(uint64_cu *p, int n) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  while (idx < n) {
    // hash address
    p[idx] = hash((uint64_cu)&p[idx]);
    idx += blockDim.x * gridDim.x;
  }
}

__host__ void printBits(uint64_cu *x) {
  std::bitset<sizeof(uint64_cu) * CHAR_BIT> b(*x);
  std::cout << b << std::endl;
}

int main(void) {
  int numIndexes = 950000000;
  int k = 10;

  int blockSize = 256;
  int numBlocks = (numIndexes + blockSize - 1) / blockSize;

  // allocate space on host for query, k nearest distances, and k nearest
  // indexes
  uint64_cu *hostQuery;
  float *kNearestDistances;
  uint64_cu *kNearestIndexes;
  hostQuery = (uint64_cu *)malloc(sizeof(uint64_cu));
  kNearestDistances = (float *)malloc(k * sizeof(float));
  kNearestIndexes = (uint64_cu *)malloc(k * sizeof(uint64_cu));

  // allocate space to receive k nearest keys on host
  unsigned *kNearestKeys;
  kNearestKeys = (unsigned *)malloc(k * sizeof(unsigned));

  // allocate space on device for query and indexes
  uint64_cu *query, *indexes;
  hipMalloc(&query, sizeof(uint64_cu));
  hipMalloc(&indexes, numIndexes * sizeof(uint64_cu));

  unsigned *distances;
  hipMalloc(&distances, numIndexes * sizeof(unsigned));

  // allocate and initalize keys on device
  unsigned *keys;
  hipMalloc(&keys, numIndexes * sizeof(unsigned));
  thrust::sequence(thrust::device, keys, keys + numIndexes);

  // allocate working memory
  unsigned *workingMem1, *workingMem2;
  hipMalloc(&workingMem1, numIndexes * sizeof(unsigned));
  hipMalloc(&workingMem2, numIndexes * sizeof(unsigned));

  // generate random indexes on device
  randf<<<numBlocks, blockSize>>>(indexes, numIndexes);
  hipDeviceSynchronize();

  // generate random query on device and transfer to host
  randf<<<1, 1>>>(query, 1);
  hipDeviceSynchronize();
  hipMemcpy(hostQuery, query, sizeof(uint64_t), hipMemcpyDeviceToHost);

  // run and time kNearestNeighbors call
  float time;
  hipEvent_t start, stop;

  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  kNearestNeighbors(indexes, keys, query, numIndexes, k, kNearestDistances,
                    kNearestIndexes, kNearestKeys, distances, workingMem1,
                    workingMem2);

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&time, start, stop);

  printf("Execution time:  %.3f ms \n", time);

  // print results
  printf("Query: ");
  printBits(hostQuery);
  for (int i = 0; i < k; ++i) {
    printf("%d: %f ", i, kNearestDistances[i]);
    printBits(&kNearestIndexes[i]);
  }

  // free device memory
  hipFree(query);
  hipFree(indexes);
  hipFree(distances);
  hipFree(keys);
  hipFree(workingMem1);
  hipFree(workingMem2);

  // free host memory
  free(hostQuery);
  free(kNearestDistances);
  free(kNearestIndexes);
  free(kNearestKeys);

  return 0;
}