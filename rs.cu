#include "hip/hip_runtime.h"
#include <assert.h>
#include <stdio.h>
#include <stdlib.h>

#include "radix_select.cuh"

__device__ uint32_cu hash(uint32_cu a) {
  a = (a ^ 61) ^ (a >> 16);
  a = a + (a << 3);
  a = a ^ (a >> 4);
  a = a * 0x27d4eb2d;
  a = a ^ (a >> 15);
  return a;
}

__global__ void rand(int n, uint32_cu *xs) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;

  for (int i = idx; i < n; i += stride)
    xs[idx] = hash(~idx);
}

int main() {
  int n = 1 << 30;
  int k = 100;

  int blockSize = 512;
  int numBlocks = (n + blockSize - 1) / blockSize;

  // generate random numbers
  uint32_cu *xs;
  hipMalloc(&xs, n * sizeof(uint32_cu));
  rand<<<numBlocks, blockSize>>>(n, xs);
  hipDeviceSynchronize();

  // allocate and initalize keys on device
  int *keys;
  hipMalloc(&keys, n * sizeof(int));
  thrust::sequence(thrust::device, keys, keys + n);

  // allocate kSmallestKeys and kSmallestValues on host
  int *kSmallestKeys = (int *)malloc(k * sizeof(int));
  uint32_cu *kSmallestValues = (uint32_cu *)malloc(k * sizeof(int));

  // run radix select
  float time;
  hipEvent_t start, stop;

  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  uint32_cu result = radix_select(xs, keys, n, k, kSmallestValues, kSmallestKeys);

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&time, start, stop);

  printf("Execution time:  %.3f ms \n", time);
  printf("Result: %u\n", result);

  for (int i = 0; i < k; ++i) {
    printf("kSmallestKeys: %d: %d\n", i, kSmallestKeys[i]);
  }
  for (int i = 0; i < k; ++i) {
    printf("kSmallestValues: %d: %u\n", i, kSmallestValues[i]);
  }

  // // run thrust sort
  // hipEventCreate(&start);
  // hipEventCreate(&stop);
  // hipEventRecord(start, 0);

  // thrust::sort(thrust::device, xs, xs + n);
  // uint32_cu *result2 = (uint32_cu *)malloc(sizeof(uint32_cu)); 
  // hipMemcpy(result2, &xs[k - 1], sizeof(uint32_cu), hipMemcpyDeviceToHost);

  // hipEventRecord(stop, 0);
  // hipEventSynchronize(stop);
  // hipEventElapsedTime(&time, start, stop);

  // printf("Execution time:  %.3f ms \n", time);
  // printf("Result2: %u\n", *result2);

  hipFree(xs);
  hipFree(keys);

  free(kSmallestKeys);

  return 0;
}