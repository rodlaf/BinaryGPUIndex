#include "hip/hip_runtime.h"
#include <assert.h>
#include <stdio.h>
#include <stdlib.h>

#include "radix_select.cuh"

__device__ uint32_cu hash(uint32_cu a) {
  a = (a ^ 61) ^ (a >> 16);
  a = a + (a << 3);
  a = a ^ (a >> 4);
  a = a * 0x27d4eb2d;
  a = a ^ (a >> 15);
  return a;
}

__global__ void rand(int n, uint32_cu *xs) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;

  for (int i = idx; i < n; i += stride)
    xs[idx] = hash(~idx);
}

int main() {
  int n = 1 << 30;
  int k = 20000;

  int blockSize = 512;
  int numBlocks = (n + blockSize - 1) / blockSize;

  // generate random numbers
  uint32_cu *xs;
  hipMalloc(&xs, n * sizeof(uint32_cu));
  rand<<<numBlocks, blockSize>>>(n, xs);
  hipDeviceSynchronize();

  // run radix select
  float time;
  hipEvent_t start, stop;

  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  uint32_cu result = radix_select(xs, n, k);

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&time, start, stop);

  printf("Execution time:  %.3f ms \n", time);

  printf("Result: %u\n", result);

  hipFree(xs);

  return 0;
}