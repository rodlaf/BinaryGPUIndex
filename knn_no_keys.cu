#include "hip/hip_runtime.h"
#include <bitset>
#include <iostream>
#include <math.h>
#include <sys/time.h>

#include <thrust/device_vector.h>
#include <thrust/functional.h>
#include <thrust/generate.h>
#include <thrust/host_vector.h>
#include <thrust/random.h>
#include <thrust/reduce.h>

typedef unsigned long long int uint64_cu;

__device__ void cosineDistance(uint64_cu *a, uint64_cu *b, float *dest) {
  // __popcll computes the Hamming Weight of an integer (e.g., number of bits
  // that are 1)
  float a_dot_b = (float)__popcll(*a & *b);
  float a_dot_a = (float)__popcll(*a);
  float b_dot_b = (float)__popcll(*b);

  *dest = 1 - (a_dot_b / (sqrt(a_dot_a) * sqrt(b_dot_b)));
}

__global__ void computeDistances(int numIndexes, uint64_cu *query,
                                 uint64_cu *indexes, float *distances) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;

  for (int i = idx; i < numIndexes; i += stride)
    cosineDistance(query, &indexes[i], &distances[i]);
}

__host__ void printBits(uint64_cu *x) {
  std::bitset<sizeof(uint64_cu) * CHAR_BIT> b(*x);
  std::cout << b << std::endl;
}

int main(void) {
  int numIndexes = 970000000; // rough maximum on 24gb of GPU memory
  int k = 100;

  int blockSize = 256;
  int numBlocks = (numIndexes + blockSize - 1) / blockSize;

  // host memory
  uint64_cu *hostQuery;
  uint64_cu *hostIndexes;
  float *kNearestDistances;
  uint64_cu *kNearestIndexes;
  hostQuery = (uint64_cu *)malloc(sizeof(uint64_cu));
  hostIndexes = (uint64_cu *)malloc(numIndexes * sizeof(uint64_cu));
  kNearestDistances = (float *)malloc(k * sizeof(float));
  kNearestIndexes = (uint64_cu *)malloc(k * sizeof(uint64_cu));

  // device memory
  uint64_cu *query, *indexes;
  float *distances;
  hipMalloc(&query, sizeof(uint64_cu));
  hipMalloc(&indexes, numIndexes * sizeof(uint64_cu));
  hipMalloc(&distances, numIndexes * sizeof(float));

  // generate indexes on host and transfer to device
  thrust::default_random_engine rng(1234);
  thrust::uniform_int_distribution<uint64_cu> uniDist(0, UINT64_MAX);
  thrust::generate(hostIndexes, hostIndexes + numIndexes,
                   [&] { return uniDist(rng); });
  hipMemcpy(indexes, hostIndexes, numIndexes * sizeof(uint64_cu),
             hipMemcpyHostToDevice);

  // generate query on host and transfer to device
  *hostQuery = uniDist(rng);
  hipMemcpy(query, hostQuery, sizeof(uint64_cu), hipMemcpyHostToDevice);

  float time;
  hipEvent_t start, stop;

  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  // compute and retrieve k nearest neighbors of a query index
  {
    // ~23ms on ~1B indexes
    computeDistances<<<numBlocks, blockSize>>>(numIndexes, query, indexes,
                                               distances);

    // problem: needs a ton of space, contributes most to duration; no easy
    // parallelizable way to get k smallest values in an unsorted list of floats
    // ~11gb allocated on GPU by this point, needs more than double to execute 
    thrust::sort_by_key(thrust::device, distances, distances + numIndexes,
                        indexes);

    // copy k nearest distances and indexes from device to host
    hipMemcpy(kNearestDistances, distances, k * sizeof(float),
               hipMemcpyDeviceToHost);
    hipMemcpy(kNearestIndexes, indexes, k * sizeof(uint64_cu),
               hipMemcpyDeviceToHost);
  }

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&time, start, stop);

  printf("Execution time:  %.3f ms \n", time);

  // print results
  printf("Query: ");
  printBits(hostQuery);
  for (int i = 0; i < k; ++i) {
    printf("%5d: %8.8f ", i + 1, kNearestDistances[i]);
    printBits(&kNearestIndexes[i]);
  }

  // free device memory
  hipFree(query);
  hipFree(indexes);
  hipFree(distances);

  // free host memory
  free(hostQuery);
  free(hostIndexes);
  free(kNearestDistances);
  free(kNearestIndexes);

  return 0;
}