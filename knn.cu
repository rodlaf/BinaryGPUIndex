#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <sys/time.h>
#include <bitset>

#include <thrust/device_vector.h>
#include <thrust/functional.h>
#include <thrust/generate.h>
#include <thrust/host_vector.h>
#include <thrust/random.h>
#include <thrust/reduce.h>

typedef unsigned long long int uint64_cu;

__device__ void cosineSimilarity(uint64_cu *a, uint64_cu *b, float *dest) {
  // __popcll computes the Hamming Weight of an integer (e.g., number of bits
  // that are 1)
  float a_dot_b = (float)__popcll(*a & *b);
  float a_dot_a = (float)__popcll(*a);
  float b_dot_b = (float)__popcll(*b);

  *dest = a_dot_b / (sqrt(a_dot_a) * sqrt(b_dot_b));
}

__device__ void jaccardSimilarity(uint64_cu *a, uint64_cu *b, float *dest) {
  float intersectionBits = (float)__popcll(*a & *b);
  float unionBits = (float)__popcll(*a | *b);

  *dest = intersectionBits / unionBits;
}

__global__ void computeDistances(int numIndexes, uint64_cu *query,
                                  uint64_cu *indexes, float *distances) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;

  for (int i = idx; i < numIndexes; i += stride)
    cosineSimilarity(query, &indexes[i], &distances[i]);
}

__host__ void printBits(uint64_cu *x) {
  std::bitset<sizeof(uint64_cu) * CHAR_BIT> b(*x);
  std::cout << b << std::endl;
}

int main(void) {
  int numIndexes = 100;

  thrust::default_random_engine rng(1234);
  thrust::uniform_int_distribution<uint64_cu> dist(0, UINT64_MAX);

  uint64_cu *query, *indexes;
  float *distances;
  int *keys;

  hipMallocManaged(&query, sizeof(uint64_cu));
  hipMallocManaged(&indexes, numIndexes * sizeof(uint64_cu));
  hipMallocManaged(&distances, numIndexes * sizeof(float));
  hipMallocManaged(&keys, numIndexes * sizeof(int));

  thrust::generate(indexes, indexes + numIndexes, [&] { return dist(rng); });

  *query = dist(rng);

  int blockSize = 256;
  int numBlocks = (numIndexes + blockSize - 1) / blockSize;

  float time;
  hipEvent_t start, stop;

  // printf("numIndexes: %d\n", numIndexes);
  // printf("numBlocks: %d\n", numBlocks);

  // First call does some memory stuff need to think about.
  computeDistances<<<numBlocks, blockSize>>>(numIndexes, query, indexes,
                                             distances);

  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  computeDistances<<<numBlocks, blockSize>>>(numIndexes, query, indexes,
                                             distances);
  
  thrust::sequence(thrust::device, keys, keys + numIndexes);

  thrust::sort_by_key(thrust::device, distances, distances + numIndexes, keys);

  hipDeviceSynchronize();

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&time, start, stop);


  printf("Execution time:  %.3f ms \n", time);

  for (int i = 0; i < numIndexes; ++i)
    printf("%d\n", keys[i]);

  hipFree(query);
  hipFree(indexes);
  hipFree(distances);

  return 0;
}