#include "hip/hip_runtime.h"
#include <cstdio>
#include <fstream>
#include <string>
#include <unordered_map>

#include <thrust/sequence.h>

#include "kNearestNeighbors.h"

// Design decision: Separate deviceKeys and vectorKeys.
//
// vectorKeys can be very long (e.g., a whole UUID) but deviceKeys must
// be a single unsigned integer (e.g., 32 bits). This imposes a limitation
// that at most 2^32 billion vectors can be searched on a GPU at a time, no
// matter the memory constraints of the GPU. This can be changed in the future.
//
// a separate collection of deviceKey-to-vectorkey key-value pairs will be kept
// in either CPU memory or the persistent key value store (e.g., RocksDB).
//
// in addition, deviceKeys will be sequential in order to enable quick vector
// retrieval by interpreting them as indexes in the on-device vector array

// Requires keys to be sequential, representing array indexes
__global__ void retrieveVectorsFromKeys(uint64_cu *vectors, unsigned *keys,
                                        int numKeys, uint64_cu *retrieved) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;

  for (int i = idx; i < numKeys; i += stride)
    retrieved[i] = vectors[keys[i]];
}

void printBits(uint64_cu &x) {
  std::bitset<sizeof(uint64_cu) * CHAR_BIT> b(x);
  std::cout << b << std::endl;
}

using boost::uuids::random_generator;
using boost::uuids::to_string;
using boost::uuids::uuid;

class DeviceIndex {
private:
  // Pointers to device memory
  unsigned *workingMem1;
  unsigned *workingMem2;
  unsigned *workingMem3;
  uint64_cu *vectors;
  uint64_cu *deviceQueryVector;
  unsigned *deviceKeys; // sequential keys

  // Use an in-memory hash map to keep track of deviceKey to vectorKey mappings
  // In practice, this means vector ids can't be too big. An alternate 
  // implementation could retrieve ids from disk instead. This would be much 
  // slower for large k when querying
  std::unordered_map<unsigned, uuid> idMap;

public:
  int numVectors = 0;
  const char *name;

  // TODO: Make capacity public variable and ensure that it matches the passed
  // variable of the same name if a database is being reopened
  // OR rename capacity to size and make it a feature, kinda
  DeviceIndex(const char *nameParam, int capacity) {
    name = nameParam;
    
    // Allocate deviceKeys and initialize (initialization requires memory)
    hipMalloc(&deviceKeys, capacity * sizeof(unsigned));
    thrust::sequence(thrust::device, deviceKeys, deviceKeys + capacity);

    // Allocate rest of on-device memory
    hipMalloc(&workingMem1, capacity * sizeof(unsigned));
    hipMalloc(&workingMem2, capacity * sizeof(unsigned));
    hipMalloc(&workingMem3, capacity * sizeof(unsigned));
    hipMalloc(&vectors, capacity * sizeof(uint64_cu));
    hipMalloc(&deviceQueryVector, sizeof(uint64_cu));

    // Read vectors from file to device and idMap using a buffer
    int bufferSize = 4 << 20;
    uint64_cu *buffer = (uint64_cu *)malloc(bufferSize * sizeof(uint64_cu));
    int bufferCount = 0;
    auto flushBuffer = [&]() { 
      // printf("bufferCount: %d\n", bufferCount);
      hipMemcpy(vectors + numVectors, buffer, bufferCount * sizeof(uint64_cu), 
                 hipMemcpyHostToDevice);
      numVectors += bufferCount;
      bufferCount = 0;
    };

    std::ifstream f(name);
    int lineSize = sizeof(uuid) + sizeof(uint64_cu);
    assert(lineSize == 24);
    char *lineBuf = (char *) malloc(lineSize);

    int lineCount = 0;
    while (f.read(lineBuf, lineSize)) {
      lineCount++;

      // Get id and record in idMap
      uuid id;
      memcpy(&id, lineBuf, sizeof(uuid));
      idMap[numVectors + bufferCount] = id;

      // Copy vector to buffer
      memcpy(buffer + bufferCount, lineBuf + sizeof(uuid), sizeof(uint64_cu));
      bufferCount++;

      // Flush buffer to device if full
      if (bufferCount == bufferSize)
        flushBuffer();
    }
    printf("lineCount: %d\n", lineCount);

    // Flush buffer
    flushBuffer();

    free(buffer);

    printf("numVectors: %d\n", numVectors);
  }

  ~DeviceIndex() {
    hipFree(workingMem1);
    hipFree(workingMem2);
    hipFree(workingMem3);
    hipFree(vectors);
    hipFree(deviceQueryVector);
    hipFree(deviceKeys);
  }

  /*
    Inserts keys. Behaviour is undefined if ids already exist
  */
 // TODO: Increase cuda mem here so no need for capacity variable?
  void insert(int numToAdd, uuid ids[], uint64_cu vectorsToAdd[]) {
    // write ids and vectors to disk
    std::ofstream f;
    f.open(name, std::ios_base::app);
    int lineSize = sizeof(uuid) + sizeof(uint64_cu);

    char *buffer = (char *) malloc(numToAdd * lineSize);
    for (int i = 0; i < numToAdd; ++i) {
      memcpy(buffer + i * lineSize, &ids[i], 16);
      memcpy(buffer + i * lineSize + sizeof(uuid), &vectorsToAdd[i], 8);
    }
    printf("numToAdd: %d\n", numToAdd);
    f.write(buffer, numToAdd * lineSize);
    f.close();
    free(buffer);

    // insert ids into keymap
    for (int i = 0; i < numToAdd; ++i) {
      // TODO: Explain.
      idMap[numVectors + i] = ids[i];
    }

    // copy vectors to device
    hipMemcpy(vectors + numVectors, vectorsToAdd, numToAdd * sizeof(uint64_cu),
               hipMemcpyHostToDevice);

    // update numVectors
    numVectors += numToAdd;
  }

  /*

  */
  void query(uint64_cu &queryVector, int k, float kNearestDistances[],
             uint64_cu kNearestVectors[], uuid kNearestIds[]) {
    float *deviceKNearestDistances;
    unsigned *deviceKNearestKeys;
    uint64_cu *deviceKNearestVectors;
    hipMalloc(&deviceKNearestDistances, k * sizeof(float));
    hipMallocManaged(&deviceKNearestKeys, k * sizeof(unsigned));
    hipMalloc(&deviceKNearestVectors, k * sizeof(uint64_cu));

    printf("numVectors: %d\n", numVectors);

    // copy query vector to device
    hipMemcpy(deviceQueryVector, &queryVector, sizeof(uint64_cu),
               hipMemcpyHostToDevice);

    kNearestNeighbors(vectors, deviceKeys, deviceQueryVector, numVectors, k,
                      deviceKNearestDistances, deviceKNearestKeys, workingMem1,
                      workingMem2, workingMem3);

    // retrieve vectors from relevant keys
    retrieveVectorsFromKeys<<<1, 1024>>>(vectors, deviceKNearestKeys, k,
                                         deviceKNearestVectors);
    hipDeviceSynchronize();

    // copy solution from device to host specified by caller
    hipMemcpy(kNearestDistances, deviceKNearestDistances, k * sizeof(float),
               hipMemcpyDeviceToHost);
    hipMemcpy(kNearestVectors, deviceKNearestVectors, k * sizeof(uint64_cu),
               hipMemcpyDeviceToHost);
    for (int i = 0; i < k; ++i)
      kNearestIds[i] = idMap[deviceKNearestKeys[i]];

    hipFree(deviceKNearestDistances);
    hipFree(deviceKNearestKeys);
    hipFree(deviceKNearestVectors);
  }
};
