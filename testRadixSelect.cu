#include "hip/hip_runtime.h"
#include <assert.h>
#include <stdio.h>
#include <stdlib.h>

#include "radixSelect.h"

__device__ unsigned hash(unsigned a) {
  a = (a ^ 61) ^ (a >> 16);
  a = a + (a << 3);
  a = a ^ (a >> 4);
  a = a * 0x27d4eb2d;
  a = a ^ (a >> 15);
  return a;
}

__global__ void rand(int n, unsigned *xs) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;

  for (int i = idx; i < n; i += stride)
    xs[idx] = hash(~idx);
}

int main() {
  int n = 1 << 20;
  int k = 10;

  int blockSize = 512;
  int numBlocks = (n + blockSize - 1) / blockSize;

  // generate random numbers
  unsigned *xs;
  hipMalloc(&xs, n * sizeof(unsigned));
  rand<<<numBlocks, blockSize>>>(n, xs);
  hipDeviceSynchronize();

  // allocate and initalize keys on device
  unsigned *keys;
  hipMalloc(&keys, n * sizeof(unsigned));
  thrust::sequence(thrust::device, keys, keys + n);

  // allocate kSmallestKeys and kSmallestValues on device
  unsigned *kSmallestKeys;
  unsigned *kSmallestValues;
  hipMalloc(&kSmallestKeys, k * sizeof(unsigned));
  hipMalloc(&kSmallestValues, k * sizeof(unsigned));

  unsigned *tempValues1, *tempValues2;
  hipMalloc(&tempValues1, n * sizeof(unsigned));
  hipMalloc(&tempValues2, n * sizeof(unsigned));

  // allocate hostKSmallestKeys and hostKSmallestValues on host
  unsigned *hostKSmallestKeys;
  unsigned *hostKSmallestValues;
  hostKSmallestKeys = (unsigned *)malloc(k * sizeof(unsigned));
  hostKSmallestValues = (unsigned *)malloc(k * sizeof(unsigned));

  // run radix select
  float time;
  hipEvent_t start, stop;

  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  radixSelect(xs, keys, n, k, kSmallestValues, kSmallestKeys, tempValues1, tempValues2);

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&time, start, stop);

  printf("Execution time:  %.3f ms \n", time);

  // copy solution from device to host
  hipMemcpy(hostKSmallestKeys, kSmallestKeys, k * sizeof(unsigned), hipMemcpyDeviceToHost);
  hipMemcpy(hostKSmallestValues, kSmallestValues, k * sizeof(unsigned), hipMemcpyDeviceToHost);

  for (int i = 0; i < k; ++i) {
    printf("kSmallestKeys: %d: %u\n", i, hostKSmallestKeys[i]);
  }
  for (int i = 0; i < k; ++i) {
    printf("kSmallestValues: %d: %u\n", i, hostKSmallestValues[i]);
  }

  hipFree(xs);
  hipFree(keys);
  hipFree(tempValues1);
  hipFree(tempValues2);
  hipFree(kSmallestKeys);
  hipFree(kSmallestValues);

  free(hostKSmallestKeys);
  free(hostKSmallestValues);

  return 0;
}